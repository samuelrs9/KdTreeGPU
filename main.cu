#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <stdlib.h>
#include <vector>
#include <list>
#include <math.h>
#include <iostream>
#include <iomanip>

#include "src/Gpu.h"
#include "src/KdNode.h"

using std::setprecision;
using namespace std;

//Gpu *gpu;

/* Create a simple k-d tree and print its topology for inspection. */
sint main(sint argc, char **argv)
{
	// Set the defaults then parse the input arguments.
	sint numPoints = 102400;
	sint extraPoints = 0;
	sint numDimensions = 3;
	sint numThreads = 512;
	sint numBlocks = 32;
	sint searchDistance = 0.05*RAND_MAX;
	sint maximumNumberOfNodesToPrint = 5;

	for (sint i = 1; i < argc; i++) {
		if ( 0 == strcmp(argv[i], "-n") || 0 == strcmp(argv[i], "--numPoints") ) {
			numPoints = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-x") || 0 == strcmp(argv[i], "--extraPoints") ) {
			extraPoints = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-d") || 0 == strcmp(argv[i], "--numDimensions") ) {
			numDimensions = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-t") || 0 == strcmp(argv[i], "--numThreads") ) {
			numThreads = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-b") || 0 == strcmp(argv[i], "--numBlocks") ) {
			numBlocks = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-s") || 0 == strcmp(argv[i], "--searchDistance") ) {
			searchDistance = atol(argv[++i]);
			continue;
		}
		if ( 0 == strcmp(argv[i], "-p") || 0 == strcmp(argv[i], "--maximumNodesToPrint") ) {
			maximumNumberOfNodesToPrint = atol(argv[++i]);
			continue;
		}
		cout << "Unsupported command-line argument: " <<  argv[i] << endl;
		exit(1);
	}

	sint i = maximumNumberOfNodesToPrint + numDimensions + extraPoints;
	// Declare the two-dimensional coordinates array that contains (x,y,z) coordinates.
	/*
    sint coordinates[NUM_TUPLES][DIMENSIONS] = {
    {2,3,3}, {5,4,2}, {9,6,7}, {4,7,9}, {8,1,5},
    {7,2,6}, {9,4,1}, {8,4,2}, {9,7,8}, {6,3,1},
    {3,4,5}, {1,6,8}, {9,5,3}, {2,1,3}, {8,7,6},
    {5,4,2}, {6,3,1}, {8,7,6}, {9,6,7}, {2,1,3},
    {7,2,6}, {4,7,9}, {1,6,8}, {3,4,5}, {9,4,1} };
	 */
	//  gpu = new Gpu(numThreads,numBlocks,0,numDimensions);
	Gpu::gpuSetup(2, numThreads,numBlocks,numDimensions);
	if (Gpu::getNumThreads() == 0 || Gpu::getNumBlocks() == 0) {
		cout << "KdNode Tree cannot be built with " << numThreads << " threads or " << numBlocks << " blocks." << endl;
		exit(1);
	}
	cout << "Points = " << numPoints << " dimensions = " << numDimensions << ", threads = " << numThreads << ", blocks = " << numBlocks << endl;
	
	//auto max = RAND_MAX;

	srand(0);
	KdCoord (*coordinates) = new KdCoord[numPoints*numDimensions];
	for ( i = 0; i<numPoints; i++) {
		for (sint j=0; j<numDimensions; j++) {
		
			coordinates[i*numDimensions+j] = rand();
			//cout << coordinates[i*numDimensions+j] << " ";	
			//coordinates[i*numDimensions+j] = (KdCoord)rand();
			//coordinates[i*numDimensions+j] = (j==1)? (numPoints-i) : i;
			//coordinates[i*numDimensions+j] =  i;
		}
	}

	// Imprime as coordenadas de alguns pontos
	/*
	for ( i = 0; i<10; i++) {
		cout << "point " << i << ": (";
		for (sint j=0; j<numDimensions; j++) {
			cout << coordinates[i*numDimensions+j];
			if (j<numDimensions-1) { 
				cout << ", ";
			}
		}
		cout << ")" << endl;
	}*/

	// Create the k-d tree.  First copy the data to a tuple in its kdNode.
	// also null out the gt and lt references
	// create and initialize the kdNodes array
	KdNode *kdNodes = new KdNode[numPoints];
	if (kdNodes == NULL) {
		printf("Can't allocate %d kdNodes\n", numPoints);
		exit (1);
	}

	KdNode *root = KdNode::createKdTree(kdNodes, coordinates, numDimensions, numPoints);

	// Print the k-d tree "sideways" with the root at the left.
	cout << endl;

	if (searchDistance == 0){
		return 0;
	}
	TIMER_DECLARATION();

	// read the KdTree back from GPU
	Gpu::getKdTreeResults( kdNodes,  coordinates, numPoints, numDimensions);
#define VERIFY_ON_HOST
#ifdef VERIFY_ON_HOST
	sint numberOfNodes = root->verifyKdTree( kdNodes, coordinates, numDimensions, 0);
	cout <<  "Number of nodes on host = " << numberOfNodes << endl;
#endif

	TIMER_START();	

	for (sint i = 0; i < numPoints; i++) {
	
		//cout << "----------- point " << i << "------------\n";
		//Search the k-d tree for the k-d nodes that lie within the cutoff distance of the first tuple.
		
		KdCoord* query = (KdCoord *)malloc(numDimensions * sizeof(KdCoord));
		for (sint j = 0; j < numDimensions; j++) {
			query[j] = coordinates[i*numDimensions+j];
		}

		// Imprime ponto de consulta
		/*
		cout << "point query " << i << ": (";
		for (sint j=0; j<numDimensions; j++) {
			cout << query[j];
			if (j<numDimensions-1) { 
				cout << ", ";
			}
		}
		cout << ")" << endl;
		*/
			

		// KdCoord (*query) = new KdCoord[numPoints*numDimensions];
		// for ( i = 0; i<numPoints; i++) {
		// 	for (sint j=0; j<numDimensions; j++) {
		// 		query[i*numDimensions+j] = (KdCoord)rand();
		// 	}
		// }		

		list<KdNode> kdList = root->searchKdTree(kdNodes, coordinates, query, searchDistance, numDimensions, 0);

		//cout << " --> " << kdList.size() << " nodes within " << searchDistance << " units of ";

		//KdNode::printTuple(query, numDimensions);

		//cout << " in all dimensions." << endl << endl;
		/*
		if (kdList.size() != 0) {
			cout << " --> List of k-d nodes within " << searchDistance << "-unit search distance follows:" << endl;
			cout << " ----> ";
			list<KdNode>::iterator it;
			for (it = kdList.begin(); it != kdList.end(); it++) {
				KdNode::printTupleOriginal(coordinates+it->getTuple()*numDimensions, numDimensions);
				cout << " ";
			}
			cout << endl << endl;
		}*/
	}
	TIMER_STOP(double searchTime);
	cout << "searchTime = " << fixed << setprecision(4) << searchTime << " seconds" << endl << endl;
	return 0;
}
